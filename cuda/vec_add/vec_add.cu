
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

#define N (4096*4096)
#define THREADS_PER_BLOCK 512

int main()
{
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof( int );

/* get GPU device number and name */

  int dev;
  hipDeviceProp_t deviceProp;
  hipGetDevice( &dev );
  hipGetDeviceProperties( &deviceProp, dev );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

/* allocate space for device copies of a, b, c */

  hipMalloc( (void **) &d_a, size );
  hipMalloc( (void **) &d_b, size );
  hipMalloc( (void **) &d_c, size );

/* allocate space for host copies of a, b, c and setup input values */

  a = (int *)malloc( size );
  b = (int *)malloc( size );
  c = (int *)malloc( size );

  for( int i = 0; i < N; i++ )
  {
    a[i] = b[i] = i;
    c[i] = 0;
  }

/* copy inputs to device */

  hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
  hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

/* launch the kernel on the GPU */

  add<<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

/* copy result back to host */

  hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

  int success = 1;

  for( int i = 0; i < N; i++ )
  {
    if( c[i] != a[i] + b[i] )
    {
      printf("c[%d] = %d\n",i,c[i] );
      success = 0;
      break;
    } /* end if */
  }

  if( success == 1 ) printf("PASS\n");
  else               printf("FAIL\n");

/* clean up */

  free(a);
  free(b);
  free(c);
  hipFree( d_a );
  hipFree( d_b );
  hipFree( d_c );

  hipDeviceReset();
	
  return 0;
} /* end main */
