#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 20480 // Size of the matrices (N x N)

__global__ void matrixMulKernel(float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float value = 0.0;
        for (int k = 0; k < n; ++k) {
            value += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = value;
    }
}

void matrixMul(float *A, float *B, float *C, int n) {
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc((void **)&d_A, n * n * sizeof(float));
    hipMalloc((void **)&d_B, n * n * sizeof(float));
    hipMalloc((void **)&d_C, n * n * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the kernel
    matrixMulKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, n);

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %f ms\n", milliseconds);

    // Copy result back to host
    hipMemcpy(C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    float *A = (float *)malloc(N * N * sizeof(float));
    float *B = (float *)malloc(N * N * sizeof(float));
    float *C = (float *)malloc(N * N * sizeof(float));

    // Initialize matrices A and B
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = (float)(rand() % 10);
            B[i * N + j] = (float)(rand() % 10);
            C[i * N + j] = 0; // Initialize result matrix
        }
    }

    // Perform matrix multiplication
    matrixMul(A, B, C, N);

    // Print a small part of the result for verification (optional)
    printf("Result matrix C (first 5 elements):\n");
    for (int i = 0; i < 5; i++) {
        printf("%f ", C[i]);
    }
    printf("\n");

    // Clean up
    free(A);
    free(B);
    free(C);
    return 0;
}

